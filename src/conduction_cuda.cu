
#include <hip/hip_runtime.h>
/*! \file conduction_cuda.cu
 *  \brief Function to calculate the thermal conduction between cells.*/

#ifdef CUDA
#ifdef CONDUCTION_GPU

#include<cuda.h>
#include<math.h>
#include"global.h"
#include"global_cuda.h"
#include"conduction_cuda.h"

extern texture<float, 2, cudaReadModeElementType> coolTexObj;
extern texture<float, 2, cudaReadModeElementType> heatTexObj;

/*! \fn void conduction_kernel(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, Real dt, Real gamma)
 *  \brief When passed an array of conserved variables and a timestep, adjust the energy
        of each cell according to thermal conduction. */
__global__ void conduction_kernel(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, int n_fields, Real dt, Real dx, Real dy, Real dz, Real gamma, Real kappa) {
  bool debug = false;

  if(debug)
    printf("Num ghost: %i, ", n_ghost);
  // Only allocate as much shared memory as needed;
  // int numDim = 1;
  // numDim += (ny == 1) ? 0 : 1;
  // numDim += (nz == 1) ? 0 : 1;

  // Should fix as it wastes memory
  __shared__ Real shared[TPB*4];
  // shared[tid] = temp
  // shared[TPB + tid] = flux x
  // shared[2*TPB + tid] = flux y
  // shared[3*TPB + tid] = flux z

  int n_cells = nx * ny * nz;
  int i_start, i_end, j_start, j_end, k_start, k_end;
  i_start = n_ghost;
  i_end = nx - n_ghost;
  if (ny == 1) {
    j_start = 0;
    j_end = 1;
  } else {
    j_start = n_ghost;
    j_end = ny-n_ghost;
  }
  if (nz == 1) {
    k_start = 0;
    k_end = 1;
  } else {
    k_start = n_ghost;
    k_end = nz-n_ghost;
  }
 
  Real d, E;                    // density, energy
  Real n, T;            // number density, temperature, initial temperature
  Real mu;                      // mean molecular weight
  Real vx, vy, vz, p;           // (x,y,z) velocity, pressure
  Real kd;                      // kappa * density

  mu = 0.6;
  //mu = 1.27;

  // get a global thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  int id = threadIdx.x + blockId * blockDim.x;
  int zid = id / (nx*ny);
  int yid = (id - zid*nx*ny) / nx;
  int xid = id - zid*nx*ny - yid*nx;
  // and a thread id within the block
  int tid = threadIdx.x;

  // FYI id = xid + yid*nx + zid*nx*ny
  int right_id  = (xid + 1) + yid*nx + zid*nx*ny;
  int left_id   = (xid - 1) + yid*nx + zid*nx*ny;
  int front_id  = xid + (yid + 1)*nx + zid*nx*ny;
  int back_id   = xid + (yid - 1)*nx + zid*nx*ny;
  int up_id     = xid + yid*nx + (zid + 1)*nx*ny;
  int down_id   = xid + yid*nx + (zid - 1)*nx*ny;

  int right_tid = right_id  - blockId * blockDim.x;
  int left_tid  = left_id   - blockId * blockDim.x;
  int front_tid = front_id  - blockId * blockDim.x;
  int back_tid  = back_id   - blockId * blockDim.x;
  int up_tid    = up_id     - blockId * blockDim.x;
  int down_tid  = down_id   - blockId * blockDim.x;

  if(debug)
    printf("Calculated ids, ");

  __syncthreads();

  ///////////////// Calculate Cell Temperature ////////////////
  // Ghost cells need to do this to span the stencil
  if(xid >= 0 && yid >= 0 && zid >=0 && xid < nx && yid < ny && zid < nz) {
    d  =  dev_conserved[            id];        // Density
    E  =  dev_conserved[4*n_cells + id];        // Energy
    if (E < 0.0 || E != E) return;              // Make sure thread is alive
    vx =  dev_conserved[1*n_cells + id] / d;    // Velocity X
    vy =  dev_conserved[2*n_cells + id] / d;    // Velocity Y
    vz =  dev_conserved[3*n_cells + id] / d;    // Velocity Z
    p  = (E - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0); // Pressure
    p  = fmax(p, (Real) TINY_NUMBER);           // Make sure pressure isn't too low.

    if(debug)
      printf("Line 101, ");
    // calculate the number density of the gas (in cgs)
    // n = d*DENSITY_UNIT / (mu * MP);
    n = d;
    T = p / d;
    // T = p*PRESSURE_UNIT/ (n*KB);
    shared[tid] = T;
  }
  __syncthreads();

  if(debug)
    printf("Found temperatures, ");
  
  // Calculate 1D Fluxes
  // IDK why, but it fails without constraining the left_id, though
  // that should be taken care of by constarining the xid
  if(xid >= i_start && xid <= i_end && left_tid >= 0) {
    kd = kappa * 0.5 * (d + dev_conserved[left_id]);
    if(left_tid < 0 || left_tid >= sizeof(shared)/sizeof(shared[0]))
      printf("xid: %i, left_id: %i, tid: %i, left_tid: %i, length: %i\n", xid, left_id, tid, left_tid, sizeof(shared)/sizeof(shared[0]));
    shared[TPB + tid] = kd*(shared[tid] - shared[left_tid]);
  }

  if(debug)
    printf("Line 116, ");

  // Calculate 2D Fluxes
  if(ny != 1 && yid >= j_start && yid <= i_end && back_tid >= 0) {
    kd = kappa * 0.5 * (d + dev_conserved[back_id]);
    if(back_tid < 0 || back_tid >= sizeof(shared)/sizeof(shared[0]))
      printf("xid: %i, left_id: %i, tid: %i, left_tid: %i, length: %i\n", yid, back_id, tid, back_tid, sizeof(shared)/sizeof(shared[0]));
    shared[2*TPB + tid] = kd*(shared[tid] - shared[back_tid]);
  }

  if(back_tid < 0) {
    printf("back_tid: %i\n", back_tid);
  }

  if(debug)
    printf("Line 124, ");

  // Calculate 3D Fluxes
  if(nz != 1 && zid >= k_start && zid <= k_end && down_tid > 0) {
    kd = kappa * 0.5 * (d + dev_conserved[down_id]);
    shared[3*TPB + tid] = kd*(shared[tid] - shared[down_tid]);
  }

  if(debug)
    printf("Calculated fluxes, ");

  // only threads corresponding to real cells update energy
  if (xid >= i_start && xid < i_end && yid >= j_start && yid < j_end && zid >= k_start && zid < k_end) {
    
    // Update with x flux
    dev_conserved[4*n_cells + id] += (shared[TPB + right_tid] - shared[TPB + tid])*(dt/dx);

    // Update with y flux
    if(ny != 1) {
      dev_conserved[4*n_cells + id] += (shared[2*TPB + front_tid] - shared[2*TPB + tid])*(dt/dy);
    }

    // Update with z flux
    if(nz != 1) {
      dev_conserved[4*n_cells + id] += (shared[3*TPB + up_tid] - shared[3*TPB + tid])*(dt/dz);
    }
    if(debug)
      printf("Conduction done d: %f, E: %f, T: %f, F_x: %f\n", dev_conserved[id], dev_conserved[4*n_cells + id], shared[tid], shared[TPB + tid]);
  }

  return;
}

#endif // CONDUCTION_GPU
#endif // CUDA
