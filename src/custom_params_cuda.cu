
#include <hip/hip_runtime.h>
#ifdef CUDA

// #include "global.h"
#include <stdio.h>
#include "global.h"
#include "custom_params_cuda.cuh"
#include "custom_params_cuda.h"

// Define the array of params in constant memory (declared in the .cuh)
__constant__ Real custom_params[100];

/* \fn void Copy_Custom_Params(Real* parameters)
* \brief Copy the passed parameters to constant memory on the GPU */
void Copy_Custom_Params(Real* parameters) {
    printf("Custom Parameters: [%d, %d, %d, %d, %d, ...]\n", parameters[0], parameters[1], parameters[2], parameters[3], parameters[4]);
    cudaMemcpyToSymbol(custom_params, parameters, 100*sizeof(Real));
    testFunc<<<1,1>>>();
}

__global__ void testFunc() {
    printf("Custom Parameter GPU: %f\n", custom_params[0]);
}

#endif