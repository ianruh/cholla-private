
#include <hip/hip_runtime.h>
/*! \file VL_1D_cuda.cu
 *  \brief Definitions of the cuda VL algorithm functions. */

#ifdef CUDA
#ifdef VL

#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<cuda.h>
#include"global.h"
#include"global_cuda.h"
#include"hydro_cuda.h"
#include"VL_1D_cuda.h"
#include"pcm_cuda.h"
#include"plmp_cuda.h"
#include"plmc_cuda.h"
#include"ppmp_cuda.h"
#include"ppmc_cuda.h"
#include"exact_cuda.h"
#include"roe_cuda.h"
#include"hllc_cuda.h"
#include"cooling_cuda.h"
#include"conduction_cuda.h"
#include"error_handling.h"
#include"io.h"


__global__ void Update_Conserved_Variables_1D_half(Real *dev_conserved, Real *dev_conserved_half, Real *dev_F, 
                                                   int n_cells, int n_ghost, Real dx, Real dt, Real gamma, int n_fields);



Real VL_Algorithm_1D_CUDA(Real *host_conserved0, Real *host_conserved1, int nx, int x_off, int n_ghost, Real dx, Real xbound, Real dt, int n_fields)
{
  //Here, *host_conserved contains the entire
  //set of conserved variables on the grid
  //host_conserved0 contains the values at time n
  //host_conserved1 will contain the values at time n+1

  // Initialize dt values
  Real max_dti = 0;
  #ifdef COOLING_GPU
  Real min_dt = 1e10;
  #endif  

  int n_cells = nx;
  int ny = 1;
  int nz = 1;

  // set the dimensions of the cuda grid
  ngrid = (n_cells + TPB - 1) / TPB;
  dim3 dimGrid(ngrid, 1, 1);
  dim3 dimBlock(TPB, 1, 1);

  if ( !memory_allocated ) {

    // allocate an array on the CPU to hold max_dti returned from each thread block
    host_dti_array = (Real *) malloc(ngrid*sizeof(Real));
    #ifdef COOLING_GPU
    host_dt_array = (Real *) malloc(ngrid*sizeof(Real));
    #endif
  
    // allocate memory on the GPU
    CudaSafeCall( cudaMalloc((void**)&dev_conserved, n_fields*n_cells*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&dev_conserved_half, n_fields*n_cells*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Lx, n_fields*n_cells*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Rx, n_fields*n_cells*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&F_x,   n_fields*n_cells*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&dev_dti_array, ngrid*sizeof(Real)) );
    #ifdef COOLING_GPU
    CudaSafeCall( cudaMalloc((void**)&dev_dt_array, ngrid*sizeof(Real)) );
    #endif  
    #ifdef CONDUCTION_GPU
    CudaSafeCall( cudaMalloc((void**)&dev_flux_array, nx*sizeof(Real)) );
    #endif


    #ifndef DYNAMIC_GPU_ALLOC 
    // If memory is single allocated: memory_allocated becomes true and succesive timesteps won't allocate memory.
    // If the memory is not single allocated: memory_allocated remains Null and memory is allocated every timestep.
    memory_allocated = true;
    #endif 
  }

  // copy the conserved variable array onto the GPU
  CudaSafeCall( cudaMemcpy(dev_conserved, host_conserved0, n_fields*n_cells*sizeof(Real), cudaMemcpyHostToDevice) );
  CudaCheckError();

  // Step 1: Use PCM reconstruction to put conserved variables into interface arrays
  PCM_Reconstruction_1D<<<dimGrid,dimBlock>>>(dev_conserved, Q_Lx, Q_Rx, nx, n_ghost, gama, n_fields);
  CudaCheckError();

  // Step 2: Calculate first-order upwind fluxes 
  #ifdef EXACT
  Calculate_Exact_Fluxes_CUDA<<<dimGrid,dimBlock>>>(Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0, n_fields);
  #endif
  #ifdef ROE
  Calculate_Roe_Fluxes_CUDA<<<dimGrid,dimBlock>>>(Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0, n_fields);
  #endif
  #ifdef HLLC 
  Calculate_HLLC_Fluxes_CUDA<<<dimGrid,dimBlock>>>(Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0, n_fields);
  #endif
  CudaCheckError();


  // Step 3: Update the conserved variables half a timestep 
  Update_Conserved_Variables_1D_half<<<dimGrid,dimBlock>>>(dev_conserved, dev_conserved_half, F_x, n_cells, n_ghost, dx, 0.5*dt, gama, n_fields);
  CudaCheckError();


  // Step 4: Construct left and right interface values using updated conserved variables
  #ifdef PCM
  PCM_Reconstruction_1D<<<dimGrid,dimBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, nx, n_ghost, gama, n_fields);
  #endif
  #ifdef PLMC
  PLMC_cuda<<<dimGrid,dimBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0, n_fields);
  #endif  
  #ifdef PLMP
  PLMP_cuda<<<dimGrid,dimBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0, n_fields);
  #endif
  #ifdef PPMP
  PPMP_cuda<<<dimGrid,dimBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0, n_fields);
  #endif
  #ifdef PPMC
  PPMC_cuda<<<dimGrid,dimBlock>>>(dev_conserved_half, Q_Lx, Q_Rx, nx, ny, nz, n_ghost, dx, dt, gama, 0, n_fields);
  #endif
  CudaCheckError();


  // Step 5: Calculate the fluxes again
  #ifdef EXACT
  Calculate_Exact_Fluxes_CUDA<<<dimGrid,dimBlock>>>(Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0, n_fields);
  #endif
  #ifdef ROE
  Calculate_Roe_Fluxes_CUDA<<<dimGrid,dimBlock>>>(Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0, n_fields);
  #endif
  #ifdef HLLC 
  Calculate_HLLC_Fluxes_CUDA<<<dimGrid,dimBlock>>>(Q_Lx, Q_Rx, F_x, nx, ny, nz, n_ghost, gama, 0, n_fields);
  #endif
  CudaCheckError();


  // Step 6: Update the conserved variable array
  Update_Conserved_Variables_1D<<<dimGrid,dimBlock>>>(dev_conserved, F_x, n_cells, x_off, n_ghost, dx, xbound, dt, gama, n_fields);
  CudaCheckError();
   

  #ifdef DE
  Sync_Energies_1D<<<dimGrid,dimBlock>>>(dev_conserved, nx, n_ghost, gama, n_fields);
  CudaCheckError();
  #endif    


  // Apply cooling
  #ifdef COOLING_GPU
  cooling_kernel<<<dimGrid,dimBlock>>>(dev_conserved, nx, ny, nz, n_ghost, n_fields, dt, gama, dev_dt_array);
  CudaCheckError();
  #endif

  // Thermal Conduction
  #ifdef CONDUCTION_GPU
  Real kappa = 1.0;
  calculate_heat_flux_kernel<<<dimGrid, dimBlock>>>(dev_conserved, dev_flux_array, nx, ny, nz, n_ghost, n_fields, dt, dx, 1, 1, gama, kappa);
  cudaError_t err = cudaGetLastError();
  gpuErrchk(err);
  CudaCheckError();
  cudaDeviceSynchronize();
  apply_heat_fluxes_kernel<<<dimGrid, dimBlock>>>(dev_conserved, dev_flux_array, nx, ny, nz, n_ghost, dt, dx, 1, 1);
  err = cudaGetLastError();
  gpuErrchk(err);
  CudaCheckError();
  #endif

  
  // Step 7: Calculate the next timestep
  Calc_dt_1D<<<dimGrid,dimBlock>>>(dev_conserved, n_cells, n_ghost, dx, dev_dti_array, gama);
  CudaCheckError();


  // copy the conserved variable array back to the CPU
  CudaSafeCall( cudaMemcpy(host_conserved1, dev_conserved, n_fields*n_cells*sizeof(Real), cudaMemcpyDeviceToHost) );

  // copy the dti array onto the CPU
  CudaSafeCall( cudaMemcpy(host_dti_array, dev_dti_array, ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
  // iterate through to find the maximum inverse dt for this subgrid block
  for (int i=0; i<ngrid; i++) {
    max_dti = fmax(max_dti, host_dti_array[i]);
  }
  #ifdef COOLING_GPU
  // copy the dt array from cooling onto the CPU
  CudaSafeCall( cudaMemcpy(host_dt_array, dev_dt_array, ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
  // find maximum inverse timestep from cooling time
  for (int i=0; i<ngrid; i++) {
    min_dt = fmin(min_dt, host_dt_array[i]);
  }  
  if (min_dt < C_cfl/max_dti) {
    max_dti = C_cfl/min_dt;
  }
  #endif

  #ifdef DYNAMIC_GPU_ALLOC
  // If memory is not single allocated then free the memory every timestep.
  Free_Memory_VL_1D();
  #endif


  // return the maximum inverse timestep
  return max_dti;


}

void Free_Memory_VL_1D() {

  // free the CPU memory
  free(host_dti_array);
  #ifdef COOLING_GPU
  free(host_dt_array);  
  #endif  

  // free the GPU memory
  cudaFree(dev_conserved);
  cudaFree(dev_conserved_half);
  cudaFree(Q_Lx);
  cudaFree(Q_Rx);
  cudaFree(F_x);
  cudaFree(dev_dti_array);
  #ifdef COOLING_GPU
  cudaFree(dev_dt_array);
  #endif

}

__global__ void Update_Conserved_Variables_1D_half(Real *dev_conserved, Real *dev_conserved_half, Real *dev_F, int n_cells, int n_ghost, Real dx, Real dt, Real gamma, int n_fields)
{
  int id, imo;
  Real dtodx = dt/dx;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  
  #ifdef DE
  Real d, d_inv, vx, vy, vz;
  Real vx_imo, vx_ipo, P;
  int ipo;
  #endif

  // threads corresponding all cells except outer ring of ghost cells do the calculation
  if (id > 0 && id < n_cells-1)
  {
    imo = id-1;
    #ifdef DE
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    //if (d < 0.0 || d != d) printf("Negative density before half step update.\n");
    //if (P < 0.0) printf("%d Negative pressure before half step update.\n", id);
    ipo = id+1;
    vx_imo = dev_conserved[1*n_cells + imo] / dev_conserved[imo]; 
    vx_ipo = dev_conserved[1*n_cells + ipo] / dev_conserved[ipo]; 
    #endif
    // update the conserved variable array
    dev_conserved_half[            id] = dev_conserved[            id] + dtodx * (dev_F[            imo] - dev_F[            id]);
    dev_conserved_half[  n_cells + id] = dev_conserved[  n_cells + id] + dtodx * (dev_F[  n_cells + imo] - dev_F[  n_cells + id]);
    dev_conserved_half[2*n_cells + id] = dev_conserved[2*n_cells + id] + dtodx * (dev_F[2*n_cells + imo] - dev_F[2*n_cells + id]);
    dev_conserved_half[3*n_cells + id] = dev_conserved[3*n_cells + id] + dtodx * (dev_F[3*n_cells + imo] - dev_F[3*n_cells + id]);
    dev_conserved_half[4*n_cells + id] = dev_conserved[4*n_cells + id] + dtodx * (dev_F[4*n_cells + imo] - dev_F[4*n_cells + id]);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_conserved_half[(5+i)*n_cells + id] = dev_conserved[(5+i)*n_cells + id] + dtodx * (dev_F[(5+i)*n_cells + imo] - dev_F[(5+i)*n_cells + id]);
    }
    #endif
    #ifdef DE
    dev_conserved_half[(n_fields-1)*n_cells + id] = dev_conserved[(n_fields-1)*n_cells + id] 
                                       + dtodx * (dev_F[(n_fields-1)*n_cells + imo] - dev_F[(n_fields-1)*n_cells + id])
                                       + 0.5*P*(dtodx*(vx_imo-vx_ipo));
    #endif    
  }


}





#endif //VL
#endif //CUDA
